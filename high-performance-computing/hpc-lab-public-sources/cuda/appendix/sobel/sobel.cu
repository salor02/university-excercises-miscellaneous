#include "hip/hip_runtime.h"
/*
 * BSD 2-Clause License
 * 
 * Copyright (c) 2020, Alessandro Capotondi
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * 
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * @file sobel.cu
 * @author Alessandro Capotondi
 * @date 5 May 2020
 * @brief Stencil 2d - Sobel
 * 
 * @see https://dolly.fim.unimore.it/2019/course/view.php?id=152
 */

#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <opencv2/opencv.hpp>
#include <opencv2/imgcodecs/imgcodecs.hpp>
#include <opencv2/objdetect/objdetect.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace cv;
using namespace std;

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32
#endif

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

void sobel_host(unsigned char *__restrict__ orig, unsigned char *__restrict__ out, int width, int height)
{
#pragma omp parallel for simd collapse(2)
    for (int y = 1; y < height - 1; y++)
    {
        for (int x = 1; x < width - 1; x++)
        {
            int dx = (-1 * orig[(y - 1) * width + (x - 1)]) + (-2 * orig[y * width + (x - 1)]) + (-1 * orig[(y + 1) * width + (x - 1)]) +
                     (orig[(y - 1) * width + (x + 1)]) + (2 * orig[y * width + (x + 1)]) + (orig[(y + 1) * width + (x + 1)]);
            int dy = (orig[(y - 1) * width + (x - 1)]) + (2 * orig[(y - 1) * width + x]) + (orig[(y - 1) * width + (x + 1)]) +
                     (-1 * orig[(y + 1) * width + (x - 1)]) + (-2 * orig[(y + 1) * width + x]) + (-1 * orig[(y + 1) * width + (x + 1)]);
            out[y * width + x] = sqrt((float)((dx * dx) + (dy * dy)));
        }
    }
}

__global__ void sobel_v1(unsigned char *__restrict__ orig, unsigned char *__restrict__ out, int width, int height)
{
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    if (j > 0 && i > 0 && j < width - 1 && i < height - 1)
    {
        int dx = (-1 * orig[(i - 1) * width + (j - 1)]) + (-2 * orig[i * width + (j - 1)]) + (-1 * orig[(i + 1) * width + (j - 1)]) +
                 (orig[(i - 1) * width + (j + 1)]) + (2 * orig[i * width + (j + 1)]) + (orig[(i + 1) * width + (j + 1)]);
        int dy = (orig[(i - 1) * width + (j - 1)]) + (2 * orig[(i - 1) * width + j]) + (orig[(i - 1) * width + (j + 1)]) +
                 (-1 * orig[(i + 1) * width + (j - 1)]) + (-2 * orig[(i + 1) * width + j]) + (-1 * orig[(i + 1) * width + (j + 1)]);
        out[i * width + j] = sqrt((float)((dx * dx) + (dy * dy)));
    }
}

__global__ void sobel_v2(unsigned char *__restrict__ orig, unsigned char *__restrict__ out, int width, int height)
{
    //TODO Declare i and j: global output indexes
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    //TODO Declare it and jt: Thread row and column of output matrix
    int it = threadIdx.y;
    int jt = threadIdx.x;

    //TODO Declare shared input patch
    __shared__ unsigned char s_in[BLOCK_SIZE][BLOCK_SIZE];

    //TODO Load input patch
    // Each thread loads one element of the patch
    s_in[it][jt] = orig[i * width + j];

    //TODO Synchronize to make sure the sub-matrices are loaded
    // before starting the computation
    __syncthreads();

    //TODO if block boundary do
    if (jt > 0 && it > 0 && jt < BLOCK_SIZE - 1 && it < BLOCK_SIZE - 1 && j > 0 && i > 0 && j < width - 1 && i < height - 1)
    {
        int dx = (-1 * s_in[it - 1][jt - 1]) + (-2 * s_in[it][jt - 1]) + (-1 * s_in[it + 1][jt - 1]) +
                 (s_in[it - 1][jt + 1]) + (2 * s_in[it][jt + 1]) + (s_in[it + 1][jt + 1]);
        int dy = (s_in[it - 1][jt - 1]) + (2 * s_in[it - 1][jt]) + (s_in[it - 1][jt + 1]) +
                 (-1 * s_in[it + 1][jt - 1]) + (-2 * s_in[it + 1][jt]) + (-1 * s_in[it + 1][jt + 1]);
        out[i * width + j] = sqrt((float)((dx * dx) + (dy * dy)));
    }
    else if (j > 0 && i > 0 && j < width - 1 && i < height - 1)
    {
        //TODO if not-block boundary do (tip check global boundaries)
        int dx = (-1 * orig[(i - 1) * width + (j - 1)]) + (-2 * orig[i * width + (j - 1)]) + (-1 * orig[(i + 1) * width + (j - 1)]) +
                 (orig[(i - 1) * width + (j + 1)]) + (2 * orig[i * width + (j + 1)]) + (orig[(i + 1) * width + (j + 1)]);
        int dy = (orig[(i - 1) * width + (j - 1)]) + (2 * orig[(i - 1) * width + j]) + (orig[(i - 1) * width + (j + 1)]) +
                 (-1 * orig[(i + 1) * width + (j - 1)]) + (-2 * orig[(i + 1) * width + j]) + (-1 * orig[(i + 1) * width + (j + 1)]);
        out[i * width + j] = sqrt((float)((dx * dx) + (dy * dy)));
    }
}

__global__ void sobel_v3(unsigned char *__restrict__ orig, unsigned char *__restrict__ out, int width, int height)
{
    //TODO Declare i and j: global output indexes (tip: use BLOCK_SIZE-2)
    int i = threadIdx.y + blockIdx.y * (BLOCK_SIZE - 2);
    int j = threadIdx.x + blockIdx.x * (BLOCK_SIZE - 2);

    //TODO Declare it and jt: Thread row and column of output matrix
    int it = threadIdx.y;
    int jt = threadIdx.x;

    //TODO Check if i and j are out of memory
    if (i >= width && j >= height)
        return;

    //TODO Declare shared input patch
    __shared__ unsigned char s_in[BLOCK_SIZE][BLOCK_SIZE];

    //TODO Load input patch
    // Each thread loads one element of the patch
    s_in[it][jt] = orig[i * width + j];

    //TODO Synchronize to make sure the sub-matrices are loaded
    // before starting the computation
    __syncthreads();

    //TODO Update block and bound checks
    if (jt > 0 && it > 0 && jt < BLOCK_SIZE - 1 && it < BLOCK_SIZE - 1 && j > 0 && i > 0 && j < width - 1 && i < height - 1)
    {
        int dx = (-1 * s_in[it - 1][jt - 1]) + (-2 * s_in[it][jt - 1]) + (-1 * s_in[it + 1][jt - 1]) +
                 (s_in[it - 1][jt + 1]) + (2 * s_in[it][jt + 1]) + (s_in[it + 1][jt + 1]);
        int dy = (s_in[it - 1][jt - 1]) + (2 * s_in[it - 1][jt]) + (s_in[it - 1][jt + 1]) +
                 (-1 * s_in[it + 1][jt - 1]) + (-2 * s_in[it + 1][jt]) + (-1 * s_in[it + 1][jt + 1]);
        out[i * width + j] = sqrt((float)((dx * dx) + (dy * dy)));
    }
}

__global__ void sobel_v4(unsigned char *__restrict__ orig, unsigned char *__restrict__ out, int width, int height)
{
    //TODO Declare i and j: global output indexes (tip: use BLOCK_SIZE)
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    //TODO Declare it and jt: Thread row and column of output matrix
    int it = threadIdx.y;
    int jt = threadIdx.x;

    //TODO Declare shared input patch (tip: use BLOCK_SIZE+2)
    __shared__ unsigned char s_in[BLOCK_SIZE + 32][BLOCK_SIZE + 32];

    //TODO Load input patch
    // Each thread loads one element of the patch
    s_in[it][jt] = orig[i * width + j];

    //TODO Check condition and load remaining elements
    if ((it + BLOCK_SIZE) < BLOCK_SIZE + 2 && (jt) < BLOCK_SIZE + 2 && (i + BLOCK_SIZE) < width && (j) < height)
        s_in[it + BLOCK_SIZE][jt] = orig[(i + BLOCK_SIZE) * width + j];

    if ((it) < BLOCK_SIZE + 2 && (jt + BLOCK_SIZE) < BLOCK_SIZE + 2 && (i) < width && (j + BLOCK_SIZE) < height)
        s_in[it][jt + BLOCK_SIZE] = orig[i * width + j + BLOCK_SIZE];

    if ((it + BLOCK_SIZE) < BLOCK_SIZE + 2 && (jt + BLOCK_SIZE) < BLOCK_SIZE + 2 && (i + BLOCK_SIZE) < width && (j + BLOCK_SIZE) < height)
        s_in[it + BLOCK_SIZE][jt + BLOCK_SIZE] = orig[(i + BLOCK_SIZE) * width + j + BLOCK_SIZE];

    //TODO Synchronize to make sure the sub-matrices are loaded
    // before starting the computation
    __syncthreads();

    //TODO Update all idx adding y +1 and x +1
    if (jt < BLOCK_SIZE && it < BLOCK_SIZE && j < (width - 2) && i < (height - 2))
    {
        int dx = (-1 * s_in[it - 1 + 1][jt - 1 + 1]) + (-2 * s_in[it + 1][jt - 1 + 1]) + (-1 * s_in[it + 1 + 1][jt - 1 + 1]) +
                 (s_in[it - 1 + 1][jt + 1 + 1]) + (2 * s_in[it + 1][jt + 1 + 1]) + (s_in[it + 1 + 1][jt + 1 + 1]);
        int dy = (s_in[it - 1 + 1][jt - 1 + 1]) + (2 * s_in[it - 1 + 1][jt + 1]) + (s_in[it - 1 + 1][jt + 1 + 1]) +
                 (-1 * s_in[it + 1 + 1][jt - 1 + 1]) + (-2 * s_in[it + 1 + 1][jt + 1]) + (-1 * s_in[it + 1 + 1][jt + 1 + 1]);
        out[(i + 1) * width + j + 1] = sqrt((float)((dx * dx) + (dy * dy)));
    }
}

int main(int argc, char *argv[])
{
    int iret = 0;
    struct timespec rt[2];
    double wt; // walltime
    string filename("../data/buzz.jpg");

    if (argc > 1)
        filename = argv[1];

    // Load Image
    Mat image = imread(filename, IMREAD_GRAYSCALE);
    if (!image.data)
    {
        cout << "Could not open or find the image" << std::endl;
        return -1;
    }
    int width = image.size().width;
    int height = image.size().height;

    // Create Output Images
    Mat out1 = image.clone();
    Mat out2 = image.clone();
    Mat result = image.clone();
    memset(out1.ptr(), 0, sizeof(unsigned char) * width * height);
    memset(out2.ptr(), 0, sizeof(unsigned char) * width * height);
    memset(result.ptr(), 0, sizeof(unsigned char) * width * height);

    // Compute CPU Version - Golden Model
    clock_gettime(CLOCK_REALTIME, rt + 0);
    sobel_host(image.ptr(), out1.ptr(), width, height);
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("Sobel (Host) : %9.6f sec\n", wt);

    //CUDA Buffer Allocation
    unsigned char *d_image_in;
    unsigned char *d_image_out;
    gpuErrchk(hipMalloc((void **)&d_image_in, sizeof(unsigned char) * width * height));
    gpuErrchk(hipMalloc((void **)&d_image_out, sizeof(unsigned char) * width * height));
    gpuErrchk(hipMemset(d_image_out, 0, sizeof(unsigned char) * width * height));

    clock_gettime(CLOCK_REALTIME, rt + 0);
    gpuErrchk(hipMemcpy(d_image_in, image.ptr(), sizeof(unsigned char) * width * height, hipMemcpyHostToDevice));
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);
    sobel_v1<<<dimGrid, dimBlock>>>(d_image_in, d_image_out, width, height);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(out2.ptr(), d_image_out, sizeof(unsigned char) * width * height, hipMemcpyDeviceToHost));
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("Sobel-v1 (GPU) : %9.6f sec\n", wt);

    //Check results
    absdiff(out1, out2, result);
    int percentage = countNonZero(result);

    //Reset Output image
    memset(out2.ptr(), 0, sizeof(unsigned char) * width * height);
    gpuErrchk(hipMemset(d_image_out, 0, sizeof(unsigned char) * width * height));

    clock_gettime(CLOCK_REALTIME, rt + 0);
    gpuErrchk(hipMemcpy(d_image_in, image.ptr(), sizeof(unsigned char) * width * height, hipMemcpyHostToDevice));
    // dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    // dim3 dimGrid((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);
    sobel_v2<<<dimGrid, dimBlock>>>(d_image_in, d_image_out, width, height);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(out2.ptr(), d_image_out, sizeof(unsigned char) * width * height, hipMemcpyDeviceToHost));
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("Sobel-v2 (GPU) : %9.6f sec\n", wt);

    //Check results
    absdiff(out1, out2, result);
    percentage = countNonZero(result);
    if (percentage)
    {
        printf("Divergence %d\n", percentage);
        imshow("Output GPU", out2);
        imshow("error diff", result);
        waitKey(0);
    }
    assert(percentage == 0);

    //Reset Output image
    memset(out2.ptr(), 0, sizeof(unsigned char) * width * height);
    gpuErrchk(hipMemset(d_image_out, 0, sizeof(unsigned char) * width * height));

    clock_gettime(CLOCK_REALTIME, rt + 0);
    gpuErrchk(hipMemcpy(d_image_in, image.ptr(), sizeof(unsigned char) * width * height, hipMemcpyHostToDevice));
    //TODO define dimGrid, dimBlock
    //TODO add sobel_v4 call
    dim3 dimBlock_v3(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid_v3((width + (BLOCK_SIZE - 2) - 1) / (BLOCK_SIZE - 2), (height + (BLOCK_SIZE - 2) - 1) / (BLOCK_SIZE - 2));
    sobel_v3<<<dimGrid_v3, dimBlock_v3>>>(d_image_in, d_image_out, width, height);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(out2.ptr(), d_image_out, sizeof(unsigned char) * width * height, hipMemcpyDeviceToHost));
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("Sobel-v3 (GPU) : %9.6f sec\n", wt);

    //Check results
    absdiff(out1, out2, result);
    percentage = countNonZero(result);
    if (percentage)
    {
        printf("Divergence %d\n", percentage);
        imshow("Output GPU", out2);
        imshow("error diff", result);
        waitKey(0);
    }
    assert(percentage == 0);

    //Reset Output image
    memset(out2.ptr(), 0, sizeof(unsigned char) * width * height);
    gpuErrchk(hipMemset(d_image_out, 0, sizeof(unsigned char) * width * height));

    clock_gettime(CLOCK_REALTIME, rt + 0);
    gpuErrchk(hipMemcpy(d_image_in, image.ptr(), sizeof(unsigned char) * width * height, hipMemcpyHostToDevice));
    //TODO define dimGrid, dimBlock
    //TODO add sobel_v4 call
    sobel_v4<<<dimGrid, dimBlock>>>(d_image_in, d_image_out, width, height);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(out2.ptr(), d_image_out, sizeof(unsigned char) * width * height, hipMemcpyDeviceToHost));
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("Sobel-v4 (GPU) : %9.6f sec\n", wt);

    //Check results
    absdiff(out1, out2, result);
    percentage = countNonZero(result);
    if (percentage)
    {
        printf("Divergence %d\n", percentage);
        imshow("Output GPU", out2);
        imshow("error diff", result);
        waitKey(0);
    }
    assert(percentage == 0);

    gpuErrchk(hipFree(d_image_out));
    gpuErrchk(hipFree(d_image_in));

    return iret;
}
