/*
 * BSD 2-Clause License
 * 
 * Copyright (c) 2020, Alessandro Capotondi
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * 
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * @file saxpy.c
 * @author Alessandro Capotondi
 * @date 12 May 2020
 * @brief Saxpy
 * 
 * @see https://dolly.fim.unimore.it/2019/course/view.php?id=152
 */

#include <assert.h>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

#define TWO02 (1 << 2)
#define TWO04 (1 << 4)
#define TWO08 (1 << 8)
#ifndef N
#define N (1 << 27)
#endif

#ifndef BLOCK_SIZE
#define BLOCK_SIZE (512)
#endif

/*
 *SAXPY (host implementation)
 * y := a * x + y
 */
void host_saxpy(float * __restrict__ y, float a, float * __restrict__ x, int n)
{
#pragma omp parallel for simd schedule(simd: static)
    for (int i = 0; i < n; i++)
    {
        y[i] = a * x[i] + y[i];
    }
}

__global__ void gpu_saxpy(float * __restrict__ y, float a, float * __restrict__ x, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        y[i] = a * x[i] + y[i];
}

int main(int argc, const char **argv)
{
    int iret = 0;
    int n = N;
    float *h_x, *d_x;
    float *h_y, *d_y;
    float *h_z;
    float a = 101.0f / TWO02,
          b, c;
    struct timespec rt[2];
    double wt; // walltime

    if (argc > 1)
        n = atoi(argv[1]);

    //TODO Update malloc to cudaMallocHost or cudaMallocManaged (if necessary)
    if (NULL == (h_x = (float *)malloc(sizeof(float) * n)))
    {
        printf("error: memory allocation for 'x'\n");
        iret = -1;
    }
    //TODO Update malloc to cudaMallocHost or cudaMallocManaged (if necessary)
    if (NULL == (h_y = (float *)malloc(sizeof(float) * n)))
    {
        printf("error: memory allocation for 'y'\n");
        iret = -1;
    }
    if (NULL == (h_z = (float *)malloc(sizeof(float) * n)))
    {
        printf("error: memory allocation for 'z'\n");
        iret = -1;
    }
    if (0 != iret)
    {
        //TODO Update cudaFreeHost or cudaFree (if necessary)
        free(h_x);
        //TODO Update cudaFreeHost or cudaFree (if necessary)
        free(h_y);
        free(h_z);
        exit(EXIT_FAILURE);
    }

    //Init Data
    b = rand() % TWO04;
    c = rand() % TWO08;
    for (int i = 0; i < n; i++)
    {
        h_x[i] = b / (float)TWO02;
        h_y[i] = h_z[i] = c / (float)TWO04;
    }

    //TODO Remove if unecessary
    gpuErrchk(hipMalloc((void **)&d_x, sizeof(float) * n));
    gpuErrchk(hipMalloc((void **)&d_y, sizeof(float) * n));

    clock_gettime(CLOCK_REALTIME, rt + 0);
    //TODO Remove if unecessary
    gpuErrchk(hipMemcpy(d_x, h_x, sizeof(float) * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_y, h_y, sizeof(float) * n, hipMemcpyHostToDevice));
    gpu_saxpy<<<((n + BLOCK_SIZE - 1) / BLOCK_SIZE), BLOCK_SIZE>>>(d_y, a, d_x, n);
    gpuErrchk(hipPeekAtLastError());
    //TODO Remove if unecessary
    gpuErrchk(hipMemcpy(h_y, d_y, sizeof(float) * n, hipMemcpyDeviceToHost));
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("saxpy (GPU): %9.3f sec %9.1f GFLOPS\n", wt, 2 * n / wt);

    //Check Matematical Consistency
    clock_gettime(CLOCK_REALTIME, rt + 0);
    host_saxpy(h_z, a, h_x, n);
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("saxpy (Host): %9.3f sec %9.1f GFLOPS\n", wt, 2 * n / wt);
    for (int i = 0; i < n; ++i)
    {
        iret = *(int *)(h_y + i) ^ *(int *)(h_z + i);
        assert(iret == 0);
    }

    //TODO Update cudaFreeHost or cudaFree (if necessary)
    free(h_x);
    //TODO Remove if unecessary
    gpuErrchk(hipFree(d_x));
    //TODO Update cudaFreeHost or cudaFree (if necessary)
    free(h_y);
    //TODO Remove if unecessary
    gpuErrchk(hipFree(d_y));
    free(h_z);

    // CUDA exit -- needed to flush printf write buffer
    hipDeviceReset();
    return 0;
}
