#include "hip/hip_runtime.h"
/*
 * BSD 2-Clause License
 * 
 * Copyright (c) 2020, Alessandro Capotondi
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * 
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * @file exercise1.cu
 * @author Alessandro Capotondi
 * @date 27 Mar 2020
 * @brief Exercise 1
 * 
 * @see https://dolly.fim.unimore.it/2019/course/view.php?id=152
 */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>
#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

extern "C"
{
#include "utils.h"
}

#define TWO02 (1 << 2)
#define TWO04 (1 << 4)
#define TWO08 (1 << 8)
#ifndef N
#define N (1LL << 28)
#endif
#ifndef BLOCK_SIZE
#define BLOCK_SIZE (1024)
#endif

/**
 * @brief  EX 1 - Offset and Strided Accesses
 *
 * a) Measure the bandwidth accessing the memory using an offset = {1,2,4,8,16,32} (mem_update v1)
 * b) Measure the bandwidth accessing the memory using a stride = {1,2,4,8,16,32} (mem_update v2)
 * 
 * @return void
 */

#ifndef STRIDE
#define STRIDE 0
#endif

// mem_update v1 - Offseted Accesses
__global__ void mem_udpate(float * __restrict__ y, float a)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    y[(i+STRIDE)%N] = a;
}

// mem_update v2 - Strided Accesses
// __global__ void mem_udpate(float * __restrict__ y, float a)
// {
//     int i = threadIdx.x + blockIdx.x * blockDim.x;
//     y[(i*STRIDE)%N] = a;
// }

int main(int argc, const char **argv)
{
    int iret = 0;
    float *h_y, *d_y;
    float a = 101.0f / TWO02;
    
    if (NULL == (h_y = (float *)malloc(sizeof(float) * N)))
    {
        printf("error: memory allocation for 'y'\n");
        iret = -1;
    }
    if (0 != iret)
    {
        free(h_y);
        exit(EXIT_FAILURE);
    }

    //CUDA Buffer Allocation
    gpuErrchk(hipMalloc((void **)&d_y, sizeof(float) * N));
    gpuErrchk(hipMemcpy(d_y, h_y, sizeof(float) * N, hipMemcpyHostToDevice));

    start_timer();
    mem_udpate<<<128*BLOCK_SIZE,BLOCK_SIZE>>>(d_y, a);
    gpuErrchk(hipPeekAtLastError());
    hipDeviceSynchronize();
    stop_timer();
    
    gpuErrchk(hipMemcpy(h_y, d_y, sizeof(float) * N, hipMemcpyDeviceToHost));
    printf("mem_udpate (GPU): %9.3f sec %9.1f MB/s\n", elapsed_ns() / 1.0e9, (4 * 128*BLOCK_SIZE*BLOCK_SIZE) / ((1.0e6 / 1e9) * elapsed_ns()));

    //CUDA Buffer Allocation
    free(h_y);
    gpuErrchk(hipFree(d_y));

    // CUDA exit -- needed to flush printf write buffer
    hipDeviceReset();
    return 0;
}
