#include "hip/hip_runtime.h"
/*
 * BSD 2-Clause License
 * 
 * Copyright (c) 2020, Alessandro Capotondi
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * 
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * @file exercise2.cu
 * @author Alessandro Capotondi
 * @date 5 May 2020
 * @brief Exercise 2 - CUDA MATMUL Optimized
 * 
 * @see https://dolly.fim.unimore.it/2019/course/view.php?id=152
 */

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

extern "C"
{
#include "utils.h"
}

#define TWO02 (1 << 2)
#define TWO04 (1 << 4)
#define TWO08 (1 << 8)

#ifndef N
#define N (1 << 10)
#endif
#ifndef TILE_W
#define TILE_W 128
#endif
#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32
#endif

void gemm(float *__restrict__ a, float *__restrict__ b, float *__restrict__ c, int n)
{

#pragma omp parallel for collapse(2)
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            float sum = 0.0;
            for (int k = 0; k < n; ++k)
            {
                sum += a[i * n + k] * b[k * n + j];
            }
            c[i * n + j] = sum;
        }
    }
}

__global__ void gemm_v1(float *__restrict__ a, float *__restrict__ b, float *__restrict__ c, int n)
{
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;

    float sum = 0.0;
    for (int k = 0; k < n; ++k)
    {
        sum += a[row * n + k] * b[k * n + col];
    }
    c[row * n + col] = sum;
}

__global__ void gemm_v2(float *__restrict__ a, float *__restrict__ b, float *__restrict__ c, int n)
{
    //TODO Shared memory used to store Asub and Bsub respectively

    //TODO Block row and column

    //TODO Thread row and column within Csub

    //TODO Each thread computes one element of Csub
    // by accumulating results into Cvalue

    //TODO Loop over all the sub-matrices of A and B that are
    // required to compute Csub.
    // Multiply each pair of sub-matrices together
    // and accumulate the results.
    for (int kb = 0; kb < (n / BLOCK_SIZE); ++kb)
    {
        //TODO Get the starting address (a_offset) of Asub
        // (sub-matrix of A of dimension BLOCK_SIZE x BLOCK_SIZE)
        // Asub is located i_block sub-matrices to the right and
        // k_block sub-matrices down from the upper-left corner of A
        //TODO Get the starting address (b_offset) of Bsub

        //TODO Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix

        //TODO Synchronize to make sure the sub-matrices are loaded
        // before starting the computation

        //TODO Multiply As and Bs together

        //TODO Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
    }

    //TODO Each thread block computes one sub-matrix Csub of C
}

__global__ void gemm_v3(float *__restrict__ a, float *__restrict__ b, float *__restrict__ c, int n)
{
    //TODO Shared memory used to store Asub and Bsub respectively

    //TODO Block row and column

    //TODO Thread row and column within Csub

    //TODO Each thread computes one element of Csub
    // by accumulating results into Cvalue

    //TODO Loop over all the sub-matrices of A and B that are
    // required to compute Csub.
    // Multiply each pair of sub-matrices together
    // and accumulate the results.
    for (int kb = 0; kb < (n / BLOCK_SIZE); ++kb)
    {
        //TODO Get the starting address (a_offset) of Asub
        // (sub-matrix of A of dimension BLOCK_SIZE x BLOCK_SIZE)
        // Asub is located i_block sub-matrices to the right and
        // k_block sub-matrices down from the upper-left corner of A
        //TODO Get the starting address (b_offset) of Bsub (Coalesced Access)

        //TODO Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix

        //TODO Synchronize to make sure the sub-matrices are loaded
        // before starting the computation

        //TODO Multiply As and Bs together

        //TODO Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
    }

    //TODO Each thread block computes one sub-matrix Csub of C
}

int main(int argc, char *argv[])
{
    int n = N, iret = 0;
    float *a, *b, *c, *g;
    struct timespec rt[2];
    double wt; // walltime

    if (argc > 1)
        n = atoi(argv[1]);

    if (NULL == (a = (float *)malloc(sizeof(*a) * n * n)))
    {
        printf("error: memory allocation for 'x'\n");
        iret = -1;
    }
    if (NULL == (b = (float *)malloc(sizeof(*b) * n * n)))
    {
        printf("error: memory allocation for 'y'\n");
        iret = -1;
    }
    if (NULL == (c = (float *)malloc(sizeof(*c) * n * n)))
    {
        printf("error: memory allocation for 'z'\n");
        iret = -1;
    }
    if (NULL == (g = (float *)malloc(sizeof(*g) * n * n)))
    {
        printf("error: memory allocation for 'z'\n");
        iret = -1;
    }

    if (0 != iret)
    {
        free(a);
        free(b);
        free(c);
        free(g);
        exit(EXIT_FAILURE);
    }

    //Init Data
    int _b = rand() % TWO04;
    int _c = rand() % TWO08;
#pragma omp parallel for
    for (int i = 0; i < n * n; i++)
    {
        a[i] = _b / (float)TWO02;
        b[i] = _c / (float)TWO04;
        c[i] = g[i] = 0.0;
    }

    clock_gettime(CLOCK_REALTIME, rt + 0);
    gemm(a, b, g, n);
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("GEMM (Host) : %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * n * n * n / (1.0e9 * wt));

    //CUDA Buffer Allocation
    float *d_a, *d_b, *d_c;
    gpuErrchk(hipMalloc((void **)&d_a, sizeof(float) * n * n));
    gpuErrchk(hipMalloc((void **)&d_b, sizeof(float) * n * n));
    gpuErrchk(hipMalloc((void **)&d_c, sizeof(float) * n * n));

    clock_gettime(CLOCK_REALTIME, rt + 0);
    gpuErrchk(hipMemcpy(d_a, a, sizeof(float) * n * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, sizeof(float) * n * n, hipMemcpyHostToDevice));
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((n + (BLOCK_SIZE)-1) / (BLOCK_SIZE), (n + (BLOCK_SIZE)-1) / (BLOCK_SIZE));
    gemm_v1<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(c, d_c, sizeof(float) * n * n, hipMemcpyDeviceToHost));
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("GEMM-v1 (GPU): %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * n * n * n / (1.0e9 * wt));

    for (int i = 0; i < n * n; i++)
    {
        iret = *(int *)(g + i) ^ *(int *)(c + i);
        assert(iret == 0);
    }

    clock_gettime(CLOCK_REALTIME, rt + 0);
    gpuErrchk(hipMemcpy(d_a, a, sizeof(float) * n * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, sizeof(float) * n * n, hipMemcpyHostToDevice));
    //dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    //dim3 dimGrid((n + (BLOCK_SIZE)-1) / (BLOCK_SIZE), (n + (BLOCK_SIZE)-1) / (BLOCK_SIZE));
    gemm_v2<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(c, d_c, sizeof(float) * n * n, hipMemcpyDeviceToHost));
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("GEMM-v2 (GPU): %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * n * n * n / (1.0e9 * wt));

    for (int i = 0; i < n * n; i++)
    {
        iret = *(int *)(g + i) ^ *(int *)(c + i);
        assert(iret == 0);
    }

    clock_gettime(CLOCK_REALTIME, rt + 0);
    gpuErrchk(hipMemcpy(d_a, a, sizeof(float) * n * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, sizeof(float) * n * n, hipMemcpyHostToDevice));
    //dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    //dim3 dimGrid((n + (BLOCK_SIZE)-1) / (BLOCK_SIZE), (n + (BLOCK_SIZE)-1) / (BLOCK_SIZE));
    gemm_v3<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(c, d_c, sizeof(float) * n * n, hipMemcpyDeviceToHost));
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("GEMM-v3 (GPU): %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * n * n * n / (1.0e9 * wt));

    for (int i = 0; i < n * n; i++)
    {
        iret = *(int *)(g + i) ^ *(int *)(c + i);
        assert(iret == 0);
    }
    free(a);
    free(b);
    free(c);
    free(g);
    gpuErrchk(hipFree(d_a));
    gpuErrchk(hipFree(d_b));
    gpuErrchk(hipFree(d_c));

    return 0;
}
