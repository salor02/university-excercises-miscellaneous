#include "hip/hip_runtime.h"
/*
 * BSD 2-Clause License
 * 
 * Copyright (c) 2020, Alessandro Capotondi
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * 
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * @file constant.cu
 * @author Alessandro Capotondi
 * @date 27 Mar 2020
 * @brief Exercise 2
 * 
 * @see https://dolly.fim.unimore.it/2019/course/view.php?id=152
 */

#include <assert.h>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define gpuErrchk(ans)                        \
{                                         \
    gpuAssert((ans), __FILE__, __LINE__); \
}
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

extern "C"
{
    #include "utils.h"
}

#define TWO02 (1 << 2)
#define TWO04 (1 << 4)
#define TWO08 (1 << 8)
#ifndef N
#define N (1 << 27)
#endif

#ifndef BLOCK_SIZE
#define BLOCK_SIZE (128)
#endif

float K[4098];
//TODO declare constant K
__constant__ float cK[4098];

/*
 * Filering 
 */
void filter(float * __restrict__ y, int n)
{
#pragma omp parallel for simd schedule(simd: static)
    for (int i = 0; i < n; i++)
    {
        y[i] = y[i] - K[i%4098];
    }
}

//TODO GPU Filter implementation
__global__ void filter_v1(float * __restrict__ y, int n)
{
}

//TODO GPU Filter implementation without constant mem
__global__ void filter_v2(float * __restrict__ y, float * __restrict__ k, int n)
{
}

int main(int argc, const char **argv)
{
    int iret = 0;
    int n = N;
    float *h_y, *d_y;
    float *h_x, *d_x, *d_k;
    float *h_z;

    if (argc > 1)
        n = atoi(argv[1]);

    if (NULL == (h_x = (float *)malloc(sizeof(float) * n)))
    {
        printf("error: memory allocation for 'x'\n");
        iret = -1;
    }
    if (NULL == (h_y = (float *)malloc(sizeof(float) * n)))
    {
        printf("error: memory allocation for 'y'\n");
        iret = -1;
    }
    if (NULL == (h_z = (float *)malloc(sizeof(float) * n)))
    {
        printf("error: memory allocation for 'z'\n");
        iret = -1;
    }
    if (0 != iret)
    {
        free(h_y);
        free(h_z);
        exit(EXIT_FAILURE);
    }

    //Init Data
    float b = rand() % TWO04;
    float c = rand() % TWO08;

    for (int i = 0; i < 4098; i++)
    {
        K[i] = b;
    }
    for (int i = 0; i < n; i++)
    {
        h_x[i] = h_y[i] = h_z[i] = c / (float)TWO04;
    }

    start_timer();
    filter(h_z, n);
    stop_timer();
    printf("Filter (Host): %9.3f sec %9.1f MFLOPS\n", elapsed_ns() / 1.0e9, n / ((1.0e6 / 1e9) * elapsed_ns()));

    //CUDA Buffer Allocation
    gpuErrchk(hipMalloc((void **)&d_y, sizeof(float) * n));
    //TODO: Load Device Constant using hipMemcpyToSymbol
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cK), K, sizeof(float)*4098));

    start_timer();
    //TODO Add Code here for calling filter_v1
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(h_y, d_y, sizeof(float) * n, hipMemcpyDeviceToHost));
    stop_timer();
    printf("Filter-v1 (GPU): %9.3f sec %9.1f MFLOPS\n", elapsed_ns() / 1.0e9, n / ((1.0e6 / 1e9) * elapsed_ns()));

    //Check Matematical Consistency
    for (int i = 0; i < n; ++i)
    {
        iret = *(int *)(h_y + i) ^ *(int *)(h_z + i);
        assert(iret == 0);
    }

    //-- No-Constant version --
    gpuErrchk(hipMalloc((void **)&d_x, sizeof(float) * n));
    gpuErrchk(hipMalloc((void **)&d_k, sizeof(float) * 4098));

    start_timer();
    //TODO Add Code here for calling filter_v2ù

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(h_x, d_x, sizeof(float) * n, hipMemcpyDeviceToHost));
    stop_timer();
    printf("Filter-v2 (GPU): %9.3f sec %9.1f MFLOPS\n", elapsed_ns() / 1.0e9, n / ((1.0e6 / 1e9) * elapsed_ns()));

    //Check Matematical Consistency
    for (int i = 0; i < n; ++i)
    {
        iret = *(int *)(h_y + i) ^ *(int *)(h_x + i);
        assert(iret == 0);
    }

    //CUDA Buffer Allocation
    free(h_x);
    gpuErrchk(hipFree(d_x));
    free(h_y);
    gpuErrchk(hipFree(d_y));
    free(h_z);
    return 0;
}
