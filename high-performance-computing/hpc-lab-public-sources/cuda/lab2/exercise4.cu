#include "hip/hip_runtime.h"
/*
 * BSD 2-Clause License
 * 
 * Copyright (c) 2020, Alessandro Capotondi
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * 
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * @file exercise4.cu
 * @author Alessandro Capotondi
 * @date 5 May 2020
 * @brief Exercise 4 - Stencil 2d - Sobel
 * 
 * @see https://dolly.fim.unimore.it/2019/course/view.php?id=152
 */

#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <opencv2/opencv.hpp>
#include <opencv2/imgcodecs/imgcodecs.hpp>
#include <opencv2/objdetect/objdetect.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace cv;
using namespace std;

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32
#endif

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

extern "C"
{
#include "utils.h"
}

void sobel_host(unsigned char *__restrict__ orig, unsigned char *__restrict__ out, int width, int height)
{
#pragma omp parallel for simd collapse(2)
    for (int y = 1; y < height - 1; y++)
    {
        for (int x = 1; x < width - 1; x++)
        {
            int dx = (-1 * orig[(y - 1) * width + (x - 1)]) + (-2 * orig[y * width + (x - 1)]) + (-1 * orig[(y + 1) * width + (x - 1)]) +
                     (orig[(y - 1) * width + (x + 1)]) + (2 * orig[y * width + (x + 1)]) + (orig[(y + 1) * width + (x + 1)]);
            int dy = (orig[(y - 1) * width + (x - 1)]) + (2 * orig[(y - 1) * width + x]) + (orig[(y - 1) * width + (x + 1)]) +
                     (-1 * orig[(y + 1) * width + (x - 1)]) + (-2 * orig[(y + 1) * width + x]) + (-1 * orig[(y + 1) * width + (x + 1)]);
            out[y * width + x] = sqrt((float)((dx * dx) + (dy * dy)));
        }
    }
}

//TODO Each thread compute one pixel out reading from global memory
__global__ void sobel_v1(unsigned char *__restrict__ orig, unsigned char *__restrict__ out, int width, int height)
{
}

#ifdef V2
//TODO Each thread compute one pixel out reading from shared memory (corner case readed from global memory)
__global__ void sobel_v2(unsigned char *__restrict__ orig, unsigned char *__restrict__ out, int width, int height)
{
    //TODO Declare i and j: global output indexes


    //TODO Declare it and jt: Thread row and column of output matrix

    //TODO Declare shared input patch

    //TODO Load input patch
    // Each thread loads one element of the patch

    //TODO Synchronize to make sure the sub-matrices are loaded
    // before starting the computation

    //TODO if block boundary do
    if (jt > 0 && it > 0 && jt < BLOCK_SIZE - 1 && it < BLOCK_SIZE - 1 && j > 0 && i > 0 && j < width - 1 && i < height - 1)
    {

    }
    else if (j > 0 && i > 0 && j < width - 1 && i < height - 1)
    {
        //TODO if not-block boundary do (tip check global boundaries)
    }
}
#endif

#ifdef V3
//TODO Each thread compute one pixel out reading from shared memory.
__global__ void sobel_v3(unsigned char *__restrict__ orig, unsigned char *__restrict__ out, int width, int height)
{
    //TODO Declare i and j: global output indexes (tip: use BLOCK_SIZE-2)

    //TODO Declare it and jt: Thread row and column of output matrix

    //TODO Check if i and j are out of memory
    if (i >= width && j >= height)
        return;

    //TODO Declare shared input patch

    //TODO Load input patch
    // Each thread loads one element of the patch

    //TODO Synchronize to make sure the sub-matrices are loaded
    // before starting the computation

    //TODO Update block and bound checks
    if (jt > 0 && it > 0 && jt < BLOCK_SIZE - 1 && it < BLOCK_SIZE - 1 && j > 0 && i > 0 && j < width - 1 && i < height - 1)
    {
    }
}
#endif

#ifdef V4
//TODO Each thread compute one pixel out reading from shared memory. Avoid thread under-usage
__global__ void sobel_v4(unsigned char *__restrict__ orig, unsigned char *__restrict__ out, int width, int height)
{
    //TODO Declare i and j: global output indexes (tip: use BLOCK_SIZE)

    //TODO Declare it and jt: Thread row and column of output matrix

    //TODO Declare shared input patch (tip: use BLOCK_SIZE+2)

    //TODO Load input patch
    // Each thread loads one element of the patch

    //TODO Check condition and load remaining elements
    if ((it + BLOCK_SIZE) < BLOCK_SIZE + 2 && (jt) < BLOCK_SIZE + 2 && (i + BLOCK_SIZE) < width && (j) < height)
        s_in[it + BLOCK_SIZE][jt] = orig[(i + BLOCK_SIZE) * width + j];

    if ((it) < BLOCK_SIZE + 2 && (jt + BLOCK_SIZE) < BLOCK_SIZE + 2 && (i) < width && (j + BLOCK_SIZE) < height)
        s_in[it][jt + BLOCK_SIZE] = orig[i * width + j + BLOCK_SIZE];

    if ((it + BLOCK_SIZE) < BLOCK_SIZE + 2 && (jt + BLOCK_SIZE) < BLOCK_SIZE + 2 && (i + BLOCK_SIZE) < width && (j + BLOCK_SIZE) < height)
        s_in[it + BLOCK_SIZE][jt + BLOCK_SIZE] = orig[(i + BLOCK_SIZE) * width + j + BLOCK_SIZE];

    //TODO Synchronize to make sure the sub-matrices are loaded
    // before starting the computation

    //TODO Update all idx adding y +1 and x +1
    if (jt < BLOCK_SIZE && it < BLOCK_SIZE && j < (width - 2) && i < (height - 2))
    {
    }
}
#endif

int main(int argc, char *argv[])
{
    int iret = 0;
    struct timespec rt[2];
    double wt; // walltime
    string filename("data/buzz.jpg");

    if (argc > 1)
        filename = argv[1];

    // Load Image
    Mat image = imread(filename, IMREAD_GRAYSCALE);
    if (!image.data)
    {
        cout << "Could not open or find the image" << std::endl;
        return -1;
    }
    int width = image.size().width;
    int height = image.size().height;

    // Create Output Images
    Mat out1 = image.clone();
    Mat out2 = image.clone();
    Mat result = image.clone();
    memset(out1.ptr(), 0, sizeof(unsigned char) * width * height);
    memset(out2.ptr(), 0, sizeof(unsigned char) * width * height);
    memset(result.ptr(), 0, sizeof(unsigned char) * width * height);

    // Compute CPU Version - Golden Model
    clock_gettime(CLOCK_REALTIME, rt + 0);
    sobel_host(image.ptr(), out1.ptr(), width, height);
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("Sobel (Host) : %9.6f sec\n", wt);

    //CUDA Buffer Allocation
    unsigned char *d_image_in;
    unsigned char *d_image_out;
    gpuErrchk(hipMalloc((void **)&d_image_in, sizeof(unsigned char) * width * height));
    gpuErrchk(hipMalloc((void **)&d_image_out, sizeof(unsigned char) * width * height));
    gpuErrchk(hipMemset(d_image_out, 0, sizeof(unsigned char) * width * height));

    clock_gettime(CLOCK_REALTIME, rt + 0);
    //TODO Copy Image to the device
    
    //TODO Define Grid and Block
    
    //TODO Launch Kernel sobel_v1

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(out2.ptr(), d_image_out, sizeof(unsigned char) * width * height, hipMemcpyDeviceToHost));
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("Sobel-v1 (GPU) : %9.6f sec\n", wt);

    //Check results
    absdiff(out1, out2, result);
    int percentage = countNonZero(result);

#ifdef V2
    //Reset Output image
    memset(out2.ptr(), 0, sizeof(unsigned char) * width * height);
    gpuErrchk(hipMemset(d_image_out, 0, sizeof(unsigned char) * width * height));

    clock_gettime(CLOCK_REALTIME, rt + 0);
    //TODO Copy Image to the device
    
    //TODO Define Grid and Block
    
    //TODO Launch Kernel sobel_v2
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("Sobel-v2 (GPU) : %9.6f sec\n", wt);

    //Check results
    absdiff(out1, out2, result);
    percentage = countNonZero(result);
    if (percentage)
    {
        printf("Divergence %d\n", percentage);
        imshow("Output GPU", out2);
        imshow("error diff", result);
        waitKey(0);
    }
    assert(percentage == 0);
#endif

#ifdef V3
    //Reset Output image
    memset(out2.ptr(), 0, sizeof(unsigned char) * width * height);
    gpuErrchk(hipMemset(d_image_out, 0, sizeof(unsigned char) * width * height));

    clock_gettime(CLOCK_REALTIME, rt + 0);
    gpuErrchk(hipMemcpy(d_image_in, image.ptr(), sizeof(unsigned char) * width * height, hipMemcpyHostToDevice));
    //TODO Copy Image to the device
    
    //TODO Define Grid and Block
    
    //TODO Launch Kernel sobel_v3
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(out2.ptr(), d_image_out, sizeof(unsigned char) * width * height, hipMemcpyDeviceToHost));
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("Sobel-v3 (GPU) : %9.6f sec\n", wt);

    //Check results
    absdiff(out1, out2, result);
    percentage = countNonZero(result);
    if (percentage)
    {
        printf("Divergence %d\n", percentage);
        imshow("Output GPU", out2);
        imshow("error diff", result);
        waitKey(0);
    }
    assert(percentage == 0);
#endif
#ifdef V4
    //Reset Output image
    memset(out2.ptr(), 0, sizeof(unsigned char) * width * height);
    gpuErrchk(hipMemset(d_image_out, 0, sizeof(unsigned char) * width * height));

    clock_gettime(CLOCK_REALTIME, rt + 0);
    //TODO Copy Image to the device
    
    //TODO Define Grid and Block
    
    //TODO Launch Kernel sobel_v4
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(out2.ptr(), d_image_out, sizeof(unsigned char) * width * height, hipMemcpyDeviceToHost));
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("Sobel-v4 (GPU) : %9.6f sec\n", wt);

    //Check results
    absdiff(out1, out2, result);
    percentage = countNonZero(result);
    if (percentage)
    {
        printf("Divergence %d\n", percentage);
        imshow("Output GPU", out2);
        imshow("error diff", result);
        waitKey(0);
    }
    assert(percentage == 0);
#endif
    gpuErrchk(hipFree(d_image_out));
    gpuErrchk(hipFree(d_image_in));

    return iret;
}
