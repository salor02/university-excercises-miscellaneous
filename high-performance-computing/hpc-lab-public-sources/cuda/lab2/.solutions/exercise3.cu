#include "hip/hip_runtime.h"
/*
 * BSD 2-Clause License
 * 
 * Copyright (c) 2020, Alessandro Capotondi
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * 
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * @file exercise3.cu
 * @author Alessandro Capotondi
 * @date 5 May 2020
 * @brief Exercise 3 - Image Luminance Histogram
 * 
 * @see https://dolly.fim.unimore.it/2019/course/view.php?id=152
 */

#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <opencv2/opencv.hpp>
#include <opencv2/imgcodecs/imgcodecs.hpp>
#include <opencv2/objdetect/objdetect.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace cv;
using namespace std;

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32
#endif

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

extern "C"
{
#include "utils.h"
}

#define NBINS 256

void hist(unsigned char *__restrict__ im, int *__restrict__ hist, int width, int height)
{
#pragma omp parallel for
    for (int i = 0; i < width * height; i++)
    {
        int val = im[i];
#pragma omp atomic
        hist[val]++;
    }
}

__global__ void hist_v1(unsigned char *__restrict__ im, int *__restrict__ hist, int width, int height)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < width && j < height)
    {
        int value;
        value = im[(j * width) + i];
        atomicAdd(&(hist[value]), 1);
        //hist[value]++;
    }
}

__global__ void hist_v2(unsigned char *__restrict__ im, int *__restrict__ hist, int width, int height)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int blockIndex = (threadIdx.y * blockDim.y) + threadIdx.x;
    __shared__ int tmpHist[NBINS];

    if (blockIndex < NBINS)
    {
        tmpHist[blockIndex] = 0;
    }
    __syncthreads();

    if (i < width && j < height)
    {
        int value;
        value = im[(j * width) + i];
        atomicAdd(&(tmpHist[value]), 1);
    }
    __syncthreads();

    if (blockIndex < NBINS)
        atomicAdd(&(hist[blockIndex]), tmpHist[blockIndex]);
}

int main(int argc, char *argv[])
{
    int iret = 0;
    struct timespec rt[2];
    double wt; // walltime
    int hist_host[NBINS], hist_gpu[NBINS];

    string filename("data/buzz.jpg");

    if (argc > 1)
        filename = argv[1];

    // Load Image
    Mat image = imread(filename, IMREAD_GRAYSCALE);
    if (!image.data)
    {
        cout << "Could not open or find the image" << std::endl;
        return -1;
    }

    int width = image.size().width;
    int height = image.size().height;

    memset(hist_host, 0, NBINS * sizeof(int));
    memset(hist_gpu, 0, NBINS * sizeof(int));

    // Compute CPU Version - Golden Model
    clock_gettime(CLOCK_REALTIME, rt + 0);
    hist(image.ptr(), hist_host, width, height);
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("Hist (Host) : %9.6f sec\n", wt);

    //CUDA Buffer Allocation
    int *d_hist_gpu;
    unsigned char *d_image;
    gpuErrchk(hipMalloc((void **)&d_hist_gpu, sizeof(int) * NBINS));
    gpuErrchk(hipMalloc((void **)&d_image, sizeof(unsigned char) * width * height));

    clock_gettime(CLOCK_REALTIME, rt + 0);
    gpuErrchk(hipMemcpy(d_image, image.ptr(), sizeof(unsigned char) * width * height, hipMemcpyHostToDevice));
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);
    hist_v1<<<dimGrid, dimBlock>>>(d_image, d_hist_gpu, width, height);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(hist_gpu, d_hist_gpu, sizeof(int) * NBINS, hipMemcpyDeviceToHost));
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("Hist (GPU) : %9.6f sec\n", wt);

    for (int i = 0; i < NBINS; i++)
    {
        iret = *(int *)(hist_host + i) ^ *(int *)(hist_gpu + i);
        assert(iret == 0);
    }
    // Reset Output
    gpuErrchk(hipMemset(d_hist_gpu, 0, NBINS * sizeof(unsigned int)));

    clock_gettime(CLOCK_REALTIME, rt + 0);
    gpuErrchk(hipMemcpy(d_image, image.ptr(), sizeof(unsigned char) * width * height, hipMemcpyHostToDevice));
    //dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    //dim3 dimGrid(width/BLOCK_SIZE, height/BLOCK_SIZE);
    hist_v2<<<dimGrid, dimBlock>>>(d_image, d_hist_gpu, width, height);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(hist_gpu, d_hist_gpu, sizeof(int) * NBINS, hipMemcpyDeviceToHost));
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("Hist-2 (GPU) : %9.6f sec\n", wt);

    for (int i = 0; i < NBINS; i++)
    {
        iret = *(int *)(hist_host + i) ^ *(int *)(hist_gpu + i);
        assert(iret == 0);
    }

    gpuErrchk(hipFree(d_hist_gpu));
    gpuErrchk(hipFree(d_image));

    return iret;
}
