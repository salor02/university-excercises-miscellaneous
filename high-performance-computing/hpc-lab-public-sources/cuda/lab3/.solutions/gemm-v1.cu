#include "hip/hip_runtime.h"
/*
 * BSD 2-Clause License
 * 
 * Copyright (c) 2020, Alessandro Capotondi
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * 
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * @file gemm.cu
 * @author Alessandro Capotondi
 * @date 12 May 2020
 * @brief GEMM Kernel
 * 
 * @see https://dolly.fim.unimore.it/2019/course/view.php?id=152
 */

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

extern "C"
{
#include "utils.h"
}

#define TWO02 (1 << 2)
#define TWO04 (1 << 4)
#define TWO08 (1 << 8)

#ifndef N
#define N (1 << 10)
#endif
#ifndef TILE_W
#define TILE_W 128
#endif
#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32
#endif

#define SM 64
static void reorder(float *__restrict__ a, float *__restrict__ b, int n)
{
    for (int i = 0; i < SM; i++)
        for (int j = 0; j < SM; j++)
            b[i * SM + j] = a[i * n + j];
}

static void mm(float *__restrict__ a, float *__restrict__ b, float *__restrict__ c, int n)
{
    for (int i = 0; i < SM; i++)
    {
        for (int k = 0; k < SM; k++)
        {
            for (int j = 0; j < SM; j++)
            {
                c[i * n + j] += a[i * n + k] * b[k * SM + j];
            }
        }
    }
}
void gemm_host(float *a, float *b, float *c, int n)
{
    int bk = n / SM;
#pragma omp parallel for collapse(3)
    for (int i = 0; i < bk; i++)
    {
        for (int j = 0; j < bk; j++)
        {
            for (int k = 0; k < bk; k++)
            {
                float b2[SM * SM];
                reorder(&b[SM * (k * n + j)], b2, n);
                mm(&a[SM * (i * n + k)], b2, &c[SM * (i * n + j)], n);
            }
        }
    }
}
__global__ void gemm(float *__restrict__ a, float *__restrict__ b, float *__restrict__ c, int n)
{
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    int ib = blockIdx.y;
    int jb = blockIdx.x;

    int it = threadIdx.y;
    int jt = threadIdx.x;

    int a_offset, b_offset, c_offset;

    float Cvalue = 0.0f;
    for (int kb = 0; kb < (n / BLOCK_SIZE); ++kb)
    {
        a_offset = ib * n * BLOCK_SIZE + kb * BLOCK_SIZE;
        b_offset = kb * n * BLOCK_SIZE + jb * BLOCK_SIZE;
        As[it][jt] = a[a_offset + it * n + jt];
        Bs[it][jt] = b[b_offset + it * n + jt];
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k)
            Cvalue += As[it][k] * Bs[k][jt];

        __syncthreads();
    }

    c_offset = ib * n * BLOCK_SIZE + jb * BLOCK_SIZE;
    c[c_offset + it * n + jt] = Cvalue;
}

int main(int argc, char *argv[])
{
    int n = N, iret = 0;
    float *a, *b, *c, *g;
    struct timespec rt[2];
    double wt; // walltime

    if (argc > 1)
        n = atoi(argv[1]);

    //TODO Update malloc to hipMallocManaged
    gpuErrchk(hipHostMalloc((void **)&a, sizeof(float) * n *n));
    gpuErrchk(hipHostMalloc((void **)&b, sizeof(float) * n *n));
    gpuErrchk(hipHostMalloc((void **)&c, sizeof(float) * n *n));
    if (NULL == (g = (float *)malloc(sizeof(*g) * n * n)))
    {
        printf("error: memory allocation for 'z'\n");
        iret = -1;
    }

    if (0 != iret)
    {
        gpuErrchk(hipHostFree(a));
        gpuErrchk(hipHostFree(b));
        gpuErrchk(hipHostFree(c));
        free(g);
        exit(EXIT_FAILURE);
    }

    //Init Data
    int _b = rand() % TWO04;
    int _c = rand() % TWO08;

#pragma omp parallel for
    for (int i = 0; i < n * n; i++)
    {
        a[i] = _b / (float)TWO02;
        b[i] = _c / (float)TWO04;
        c[i] = g[i] = 0.0;
    }

    clock_gettime(CLOCK_REALTIME, rt + 0);
    gemm_host(a, b, g, n);
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("GEMM (Host) : %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * n * n * n / (1.0e9 * wt));

    //TODO Remove if unecessary
    float *d_a, *d_b, *d_c;
    gpuErrchk(hipMalloc((void **)&d_a, sizeof(float) * n * n));
    gpuErrchk(hipMalloc((void **)&d_b, sizeof(float) * n * n));
    gpuErrchk(hipMalloc((void **)&d_c, sizeof(float) * n * n));

    clock_gettime(CLOCK_REALTIME, rt + 0);
    //TODO Remove if unecessary
    gpuErrchk(hipMemcpy(d_a, a, sizeof(float) * n * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, sizeof(float) * n * n, hipMemcpyHostToDevice));
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((n + (BLOCK_SIZE)-1) / (BLOCK_SIZE), (n + (BLOCK_SIZE)-1) / (BLOCK_SIZE));
    gemm<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);
    gpuErrchk(hipPeekAtLastError());
    //TODO Remove if unecessary
    gpuErrchk(hipMemcpy(c, d_c, sizeof(float) * n * n, hipMemcpyDeviceToHost));
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("GEMM-v1 (GPU): %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * n * n * n / (1.0e9 * wt));

    for (int i = 0; i < n * n; i++)
    {
        iret = *(int *)(g + i) ^ *(int *)(c + i);
        assert(iret == 0);
    }

    //TODO Update hipHostFree or hipFree (if necessary)
    gpuErrchk(hipHostFree(a));
    gpuErrchk(hipHostFree(b));
    gpuErrchk(hipHostFree(c));
    free(g);
    //TODO Remove if unecessary
    gpuErrchk(hipFree(d_a));
    //TODO Remove if unecessary
    gpuErrchk(hipFree(d_b));
    //TODO Remove if unecessary
    gpuErrchk(hipFree(d_c));

    return 0;
}
