#include "hip/hip_runtime.h"
/*
 * BSD 2-Clause License
 * 
 * Copyright (c) 2020, Alessandro Capotondi
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * 
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * @file sobel.cu
 * @author Alessandro Capotondi
 * @date 12 May 2020
 * @brief Sobel Filtering
 * 
 * @see https://dolly.fim.unimore.it/2019/course/view.php?id=152
 */

#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <opencv2/opencv.hpp>
#include <opencv2/imgcodecs/imgcodecs.hpp>
#include <opencv2/objdetect/objdetect.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace cv;
using namespace std;

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32
#endif

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

extern "C"
{
#include "utils.h"
}

int FILTER_HOST[3][3] = {{-1, 0, 1},
                         {-2, 0, 2},
                         {-1, 0, 1}};

void sobel_host(unsigned char *__restrict__ orig, unsigned char *__restrict__ out, int width, int height)
{
#pragma omp parallel for collapse(2)
    for (int y = 1; y < height - 1; y++)
    {
        for (int x = 1; x < width - 1; x++)
        {
            int dx = 0, dy = 0;
            for (int k = -1; k <= 1; k++)
            {
                for (int z = -1; z <= 1; z++)
                {
                    dx += FILTER_HOST[k + 1][z + 1] * orig[(y + k) * width + x + z];
                    dy += FILTER_HOST[z + 1][k + 1] * orig[(y + k) * width + x + z];
                }
            }
            out[y * width + x] = sqrt((float)((dx * dx) + (dy * dy)));
        }
    }
}

__constant__ int FILTER_GPU[3][3] = {{-1, 0, 1},
                                     {-2, 0, 2},
                                     {-1, 0, 1}};

__global__ void sobel_v1(unsigned char *__restrict__ orig, unsigned char *__restrict__ out, int width, int height)
{
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    if (j > 0 && i > 0 && j < width - 1 && i < height - 1)
    {
        int dx = 0, dy = 0;
        for (int k = -1; k <= 1; k++)
        {
            for (int z = -1; z <= 1; z++)
            {
                dx += FILTER_GPU[k + 1][z + 1] * orig[(i + k) * width + j + z];
                dy += FILTER_GPU[z + 1][k + 1] * orig[(i + k) * width + j + z];
            }
        }
        out[i * width + j] = sqrt((float)((dx * dx) + (dy * dy)));
    }
}

int main(int argc, char *argv[])
{
    int iret = 0;
    struct timespec rt[2];
    string filename("data/sample.avi");

    if (argc > 1)
        filename = argv[1];

    //Open Video Example
    VideoCapture cap(filename);
    // Check if camera opened successfully
    if (!cap.isOpened())
    {
        cout << "Error opening video stream or file" << endl;
        return -1;
    }

    int width = cap.get(CAP_PROP_FRAME_WIDTH);
    int height = cap.get(CAP_PROP_FRAME_HEIGHT);
    int nCh = 3;

    // Frame Buffers
    Mat frameRGB = Mat::zeros(height, width, CV_8UC3);
    Mat frameIn = Mat::zeros(height, width, CV_8UC1);
    Mat frameOut = Mat::zeros(height, width, CV_8UC1);

    int nFrames = 0;
    double time_cnt = 0.0;
    while (1)
    {
        bool lastFrame = cap.read(frameRGB); // read a new frame from video
        if (!lastFrame)
            break;

        cvtColor(frameRGB, frameIn, COLOR_BGR2GRAY);

        // Compute CPU Version - Golden Model
        clock_gettime(CLOCK_REALTIME, rt + 0);
        sobel_host(frameIn.ptr(), frameOut.ptr(), width, height);
        clock_gettime(CLOCK_REALTIME, rt + 1);
        time_cnt+= (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
        nFrames++;

#ifdef DISPLAY
        // Show frames
        imshow("frameIn", frameIn);
        imshow("frameOut", frameOut);
        waitKey(1);
#endif
    }
    printf("Sobel (Host) : %d frames, %9.6f s per-frame (%9.6f fps)\n", nFrames, time_cnt/nFrames, 1/(time_cnt/nFrames));

    // CUDA VERSION --------------------------------------------------
    //Open Video Example
    cap = VideoCapture(filename);
    // Check if camera opened successfully
    if (!cap.isOpened())
    {
        cout << "Error opening video stream or file" << endl;
        return -1;
    }

    unsigned char *d_image_in;
    unsigned char *d_image_out;
    gpuErrchk(hipMalloc((void **)&d_image_in, sizeof(unsigned char) * width * height));
    gpuErrchk(hipMalloc((void **)&d_image_out, sizeof(unsigned char) * width * height));
    gpuErrchk(hipMemset(d_image_out, 0, sizeof(unsigned char) * width * height));
    
    nFrames = 0;
    time_cnt = 0.0;
    while (1)
    {
        bool lastFrame = cap.read(frameRGB); // read a new frame from video
        if (!lastFrame)
            break;

        cvtColor(frameRGB, frameIn, COLOR_BGR2GRAY);

        // Compute CPU Version - Golden Model
        clock_gettime(CLOCK_REALTIME, rt + 0);
        gpuErrchk(hipMemcpy(d_image_in, frameIn.ptr(), sizeof(unsigned char) * width * height, hipMemcpyHostToDevice));
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);
        sobel_v1<<<dimGrid, dimBlock>>>(d_image_in, d_image_out, width, height);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipMemcpy(frameOut.ptr(), d_image_out, sizeof(unsigned char) * width * height, hipMemcpyDeviceToHost));
        clock_gettime(CLOCK_REALTIME, rt + 1);
        time_cnt+= (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
        nFrames++;

#ifdef DISPLAY
        // Show frames
        imshow("frameIn", frameIn);
        imshow("frameOut", frameOut);
        waitKey(1);
#endif
    }
    printf("Sobel (GPU) : %d frames, %9.6f s per-frame (%9.6f fps)\n", nFrames, time_cnt/nFrames, 1/(time_cnt/nFrames));

    gpuErrchk(hipFree(d_image_out));
    gpuErrchk(hipFree(d_image_in));
    frameOut.release();
    frameIn.release();
    frameRGB.release();
    cap.release();

    return iret;
}
