#include "hip/hip_runtime.h"
/*
 * BSD 2-Clause License
 * 
 * Copyright (c) 2020, Alessandro Capotondi
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * 
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * @file exercise3.cu
 * @author Alessandro Capotondi
 * @date 27 Mar 2020
 * @brief Exercise 3 - CUDA MATMUL
 * 
 * @see https://dolly.fim.unimore.it/2019/course/view.php?id=152
 */
 
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

extern "C"
{
#include "utils.h"
}

#define TWO02 (1 << 2)
#define TWO04 (1 << 4)
#define TWO08 (1 << 8)


#ifndef N
#define N (1 << 10)
#endif
#ifndef TILE_W
#define TILE_W 4
#endif
#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32
#endif

void gemm(float * __restrict__ a, float * __restrict__ b, float * __restrict__ c, int n)
{
    
#pragma omp parallel for collapse(2)
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            float sum = 0.0;
            for (int k = 0; k < n; ++k)
            {
                sum += a[i * n + k] * b[k *n + j];
            }
            c[i * n + j] = sum;
        }
    }
}

/**
 * @brief  EX 3 - Complete Matrix Multiplication
 */
__global__ void gemm_kernel(float * __restrict__ a, float * __restrict__ b, float * __restrict__ c, int n)
{
    int row = (blockIdx.x * blockDim.x * TILE_W) + (threadIdx.x * TILE_W);
    int col = (blockIdx.y * blockDim.y * TILE_W) + (threadIdx.y * TILE_W);
    int end_row = row+TILE_W < n ? row+TILE_W : n;
    int end_col = col+TILE_W < n ? col+TILE_W : n;

    for (int i = row; i < end_row; ++i)
    {
        for (int j = col; j < end_col; ++j)
        {
            float sum = 0.0;
            for (int k = 0; k < n; ++k)
            {
                sum += a[i * n + k] * b[k *n + j];
            }
            c[i * n + j] = sum;
        }
    }
}

int main(int argc, char *argv[])
{
    int n = N, iret = 0;
    float *a, *b, *c, *g;
    struct timespec rt[2];
    double wt; // walltime

    if (argc > 1)
        n = atoi(argv[1]);

    if (NULL == (a = (float *)malloc(sizeof(*a) * n * n)))
    {
        printf("error: memory allocation for 'x'\n");
        iret = -1;
    }
    if (NULL == (b = (float *)malloc(sizeof(*b) * n * n)))
    {
        printf("error: memory allocation for 'y'\n");
        iret = -1;
    }
    if (NULL == (c = (float *)malloc(sizeof(*c) * n * n)))
    {
        printf("error: memory allocation for 'z'\n");
        iret = -1;
    }
    if (NULL == (g = (float *)malloc(sizeof(*g) * n * n)))
    {
        printf("error: memory allocation for 'z'\n");
        iret = -1;
    }

    if (0 != iret)
    {
        free(a);
        free(b);
        free(c);
        free(g);
        exit(EXIT_FAILURE);
    }

    //Init Data
    int _b = rand() % TWO04;
    int _c = rand() % TWO08;
#pragma omp parallel for
    for (int i = 0; i < n * n; i++)
    {
        a[i] = _b / (float)TWO02;
        b[i] = _c / (float)TWO04;
        c[i] = g[i] = 0.0;
    }

    clock_gettime(CLOCK_REALTIME, rt + 0);
    gemm(a, b, g, n);
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("GEMM (Host) : %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * n * n * n / (1.0e9 * wt));

    //CUDA Buffer Allocation
    float *d_a, *d_b, *d_c;
    gpuErrchk(hipMalloc((void **)&d_a, sizeof(float) * n * n));
    gpuErrchk(hipMalloc((void **)&d_b, sizeof(float) * n * n));
    gpuErrchk(hipMalloc((void **)&d_c, sizeof(float) * n * n));

    clock_gettime(CLOCK_REALTIME, rt + 0);
    gpuErrchk(hipMemcpy(d_a, a, sizeof(float) * n * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, sizeof(float) * n * n, hipMemcpyHostToDevice));
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    dim3 dimGrid((n+(BLOCK_SIZE+TILE_W)-1)/(BLOCK_SIZE+TILE_W),(n+(BLOCK_SIZE+TILE_W)-1)/(BLOCK_SIZE+TILE_W));
    gemm_kernel<<<dimGrid, dimBlock>>> (d_a, d_b, d_c, n);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(c, d_c, sizeof(float) * n * n, hipMemcpyDeviceToHost));
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("GEMM (GPU): %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * n * n * n / (1.0e9 * wt));

    for (int i = 0; i < n * n; i++)
    {
        iret = *(int *)(g + i) ^ *(int *)(c + i);
        assert(iret == 0);
    }
    free(a);
    free(b);
    free(c);
    free(g);
    gpuErrchk(hipFree(d_a));
    gpuErrchk(hipFree(d_b));
    gpuErrchk(hipFree(d_c));

    return 0;
}
