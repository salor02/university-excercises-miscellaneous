#include "hip/hip_runtime.h"
/*
 * BSD 2-Clause License
 * 
 * Copyright (c) 2020, Alessandro Capotondi
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * 
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * @file exercise1.c
 * @author Alessandro Capotondi
 * @date 27 Mar 2020
 * @brief Exercise 2
 * 
 * @see https://dolly.fim.unimore.it/2019/course/view.php?id=152
 */

#include <assert.h>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

extern "C"
{
#include "utils.h"
}

#define TWO02 (1 << 2)
#define TWO04 (1 << 4)
#define TWO08 (1 << 8)
#ifndef N
#define N (1 << 27)
#endif

#ifndef BLOCK_SIZE
#define BLOCK_SIZE (1024)
#endif

/*
 *SAXPY (host implementation)
 * y := a * x + y
 */
void host_saxpy(float * __restrict__ y, float a, float * __restrict__ x, int n)
{
#pragma omp parallel for simd schedule(simd: static)
    for (int i = 0; i < n; i++)
    {
        y[i] = a * x[i] + y[i];
    }
}

__global__ void gpu_saxpy(float * __restrict__ y, float a, float * __restrict__ x, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        y[i] = a * x[i] + y[i];
}

int main(int argc, const char **argv)
{
    int iret = 0;
    int n = N;
    float *h_x, *d_x;
    float *h_y, *d_y;
    float *h_z;
    float a = 101.0f / TWO02,
          b, c;

    if (argc > 1)
        n = atoi(argv[1]);

    if (NULL == (h_x = (float *)malloc(sizeof(float) * n)))
    {
        printf("error: memory allocation for 'x'\n");
        iret = -1;
    }
    if (NULL == (h_y = (float *)malloc(sizeof(float) * n)))
    {
        printf("error: memory allocation for 'y'\n");
        iret = -1;
    }
    if (NULL == (h_z = (float *)malloc(sizeof(float) * n)))
    {
        printf("error: memory allocation for 'z'\n");
        iret = -1;
    }
    if (0 != iret)
    {
        free(h_x);
        free(h_y);
        free(h_z);
        exit(EXIT_FAILURE);
    }

    //Init Data
    b = rand() % TWO04;
    c = rand() % TWO08;
    for (int i = 0; i < n; i++)
    {
        h_x[i] = b / (float)TWO02;
        h_y[i] = h_z[i] = c / (float)TWO04;
    }

    //CUDA Buffer Allocation
    gpuErrchk(hipMalloc((void **)&d_x, sizeof(float) * n));
    gpuErrchk(hipMalloc((void **)&d_y, sizeof(float) * n));

    start_timer();
    gpuErrchk(hipMemcpy(d_x, h_x, sizeof(float) * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_y, h_y, sizeof(float) * n, hipMemcpyHostToDevice));
    gpu_saxpy<<<((n + BLOCK_SIZE - 1) / BLOCK_SIZE), BLOCK_SIZE>>>(d_y, a, d_x, n);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(h_y, d_y, sizeof(float) * n, hipMemcpyDeviceToHost));
    stop_timer();
    printf("saxpy (GPU): %9.3f sec %9.1f MFLOPS\n", elapsed_ns() / 1.0e9, 2 * n / ((1.0e6 / 1e9) * elapsed_ns()));

    //Check Matematical Consistency
    start_timer();
    host_saxpy(h_z, a, h_x, n);
    stop_timer();
    printf("saxpy (Host): %9.3f sec %9.1f MFLOPS\n", elapsed_ns() / 1.0e9, 2 * n / ((1.0e6 / 1e9) * elapsed_ns()));
    for (int i = 0; i < n; ++i)
    {
        iret = *(int *)(h_y + i) ^ *(int *)(h_z + i);
        assert(iret == 0);
    }

    //CUDA Buffer Allocation
    free(h_x);
    gpuErrchk(hipFree(d_x));
    free(h_y);
    gpuErrchk(hipFree(d_y));
    free(h_z);

    // CUDA exit -- needed to flush printf write buffer
    hipDeviceReset();
    return 0;
}
